#include "cudaUtil.cuh"
#include <stdio.h>

// for error-handling on operations that return hipError_t
void checkReturnedError(hipError_t error, int line)
{
  if (error != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(error), __FILE__, line);
    // hipDeviceReset();
    exit(EXIT_FAILURE);
  }
}

// for error-handling on operations that do not return any error
void checkError(int line)
{
  hipError_t error = hipGetLastError();
  checkReturnedError(error, line);
}
