#include "hip/hip_runtime.h"
#include "Sciara.h"
#include "cal2DBuffer.h"
#include "cudaUtil.cuh"

/*
  &sciara->simulation->vent                // no allocation, vector<TVent>
  &sciara->simulation->total_emitted_lava  // no allocation, double
  *sciara->substates->Sh                   // done
  *sciara->substates->Sh_next              // done
  *sciara->substates->ST                   // done
  *sciara->substates->ST-next              // done
  *sciara->substates->Sz                   // done
  *sciara->substates->Sz_next              // done
  *sciara->substates->Mf                   // done
  *sciara->substates->Mb                   // done
  *sciara->substates->Mhs                  // done
  *sciara->X->Xi                           // done
  *sciara->X->Xj                           // done
*/

void allocateSubstates(Sciara *sciara)
{
  hipError_t error = hipMallocManaged(&(sciara)->substates->Sz, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Sz");
  error = hipMallocManaged(&(sciara)->substates->Sz_next, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Sz_next");
  error = hipMallocManaged(&(sciara)->substates->Sh, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Sh");
  error = hipMallocManaged(&(sciara)->substates->Sh_next, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Sh_next");
  error = hipMallocManaged(&(sciara)->substates->ST, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->ST");
  error = hipMallocManaged(&(sciara)->substates->ST_next, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->ST_next");
  error = hipMallocManaged(&(sciara)->substates->Mf, sizeof(double) * sciara->domain->rows * sciara->domain->cols * NUMBER_OF_OUTFLOWS);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Mf");
  // sciara->substates->Mv       = new (std::nothrow)    int[sciara->domain->rows*sciara->domain->cols];
  error = hipMallocManaged(&(sciara)->substates->Mb, sizeof(bool) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Mb");
  error = hipMallocManaged(&(sciara)->substates->Mhs, sizeof(double) * sciara->domain->rows * sciara->domain->cols);
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara substates->Mhs");
}

void deallocateSubstates(Sciara *sciara)
{
  hipError_t error;
  if (sciara->substates->Sz)
  {
    error = hipFree(sciara->substates->Sz);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Sz");
  }
  if (sciara->substates->Sz_next)
  {
    error = hipFree(sciara->substates->Sz_next);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Sz_next");
  }
  if (sciara->substates->Sh)
  {
    error = hipFree(sciara->substates->Sh);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Sh");
  }
  if (sciara->substates->Sh_next)
  {
    error = hipFree(sciara->substates->Sh_next);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Sh_next");
  }
  if (sciara->substates->ST)
  {
    error = hipFree(sciara->substates->ST);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->ST");
  }
  if (sciara->substates->ST_next)
  {
    error = hipFree(sciara->substates->ST_next);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->ST_next");
  }
  if (sciara->substates->Mf)
  {
    error = hipFree(sciara->substates->Mf);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Mf");
  }
  /*
  if(sciara->substates->Mv)
  {
    delete[] sciara->substates->Mv;
  }
  */
  if (sciara->substates->Mb)
  {
    error = hipFree(sciara->substates->Mb);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Mb");
  }
  if (sciara->substates->Mhs)
  {
    error = hipFree(sciara->substates->Mhs);
    checkReturnedError(error, __LINE__, "Error deallocating memory for sciara substates->Mhs");
  }
}

void evaluatePowerLawParams(double PTvent, double PTsol, double value_sol, double value_vent, double &k1, double &k2)
{
  k2 = (log10(value_vent) - log10(value_sol)) / (PTvent - PTsol);
  k1 = log10(value_sol) - k2 * (PTsol);
}

void simulationInitialize(Sciara *sciara)
{
  // dichiarazioni
  unsigned int maximum_number_of_emissions = 0;

  // azzeramento dello step dell'AC
  sciara->simulation->step = 0;
  sciara->simulation->elapsed_time = 0;

  // determinazione numero massimo di passi
  for (unsigned int i = 0; i < sciara->simulation->emission_rate.size(); i++)
    if (maximum_number_of_emissions < sciara->simulation->emission_rate[i].size())
      maximum_number_of_emissions = sciara->simulation->emission_rate[i].size();
  // maximum_steps_from_emissions = (int)(emission_time/Pclock*maximum_number_of_emissions);
  sciara->simulation->effusion_duration = sciara->simulation->emission_time * maximum_number_of_emissions;
  sciara->simulation->total_emitted_lava = 0;

  // definisce il bordo della morfologia
  MakeBorder(sciara);

  // calcolo a b (parametri viscosità) c d (parametri resistenza al taglio)
  evaluatePowerLawParams(
      sciara->parameters->PTvent,
      sciara->parameters->PTsol,
      sciara->parameters->Pr_Tsol,
      sciara->parameters->Pr_Tvent,
      sciara->parameters->a,
      sciara->parameters->b);
  evaluatePowerLawParams(
      sciara->parameters->PTvent,
      sciara->parameters->PTsol,
      sciara->parameters->Phc_Tsol,
      sciara->parameters->Phc_Tvent,
      sciara->parameters->c,
      sciara->parameters->d);
}

int _Xi[] = {0, -1, 0, 0, 1, -1, 1, 1, -1}; // Xj: Moore neighborhood row coordinates (see below)
int _Xj[] = {0, 0, -1, 1, 0, -1, -1, 1, 1}; // Xj: Moore neighborhood col coordinates (see below)
void init(Sciara *&sciara)
{
  sciara = new Sciara;

  (sciara)->domain = new Domain;

  (sciara)->X = new NeighsRelativeCoords;

  hipError_t error = hipMallocManaged(&(sciara)->X->Xi, sizeof(int) * MOORE_NEIGHBORS);
  checkReturnedError(error, __LINE__);

  error = hipMallocManaged(&(sciara)->X->Xj, sizeof(int) * MOORE_NEIGHBORS);
  checkReturnedError(error, __LINE__);

  for (int n = 0; n < MOORE_NEIGHBORS; n++)
  {
    (sciara)->X->Xi[n] = _Xi[n];
    (sciara)->X->Xj[n] = _Xj[n];
  }

  (sciara)->substates = new Substates;
  // allocateSubstates(sciara); // substates allocation is done when the confiugration is loaded (see io.cu)
  (sciara)->parameters = new Parameters;
  (sciara)->simulation = new Simulation;
  error = hipMallocManaged(&(sciara)->simulation->vent, sizeof(vector<TVent>));
  checkReturnedError(error, __LINE__, "Error allocating memory for sciara simulation->vent");
}

void finalize(Sciara *&sciara)
{
  deallocateSubstates((sciara));
  delete (sciara)->domain;
  hipFree(sciara->X->Xi);
  hipFree(sciara->X->Xj);
  delete (sciara)->X;
  delete (sciara)->substates;
  delete (sciara)->parameters;
  delete (sciara)->simulation;
  delete sciara;
  sciara = NULL;
}

void MakeBorder(Sciara *sciara)
{
  int j, i;

  // prima riga
  i = 0;
  for (j = 0; j < sciara->domain->cols; j++)
    if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
      calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);

  // ultima riga
  i = sciara->domain->rows - 1;
  for (j = 0; j < sciara->domain->cols; j++)
    if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
      calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);

  // prima colonna
  j = 0;
  for (i = 0; i < sciara->domain->rows; i++)
    if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
      calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);

  // ultima colonna
  j = sciara->domain->cols - 1;
  for (i = 0; i < sciara->domain->rows; i++)
    if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
      calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);

  // il resto
  for (int i = 1; i < sciara->domain->rows - 1; i++)
    for (int j = 1; j < sciara->domain->cols - 1; j++)
      if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
      {
        for (int k = 1; k < MOORE_NEIGHBORS; k++)
          if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i + sciara->X->Xi[k], j + sciara->X->Xj[k]) < 0)
          {
            calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);
            break;
          }
      }
}
